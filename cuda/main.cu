
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

__device__ bool is_prime(int number);
__global__ void loop_prime(int max,int pas, int * j);
void go(const int max, const int number_block);

/*
    argv[1] = limit
    argv[2] = Number of block
*/
int main(int argc, char *argv[]) {

    time_t timeStart = time(0);

    go(atoi(argv[1]),atoi(argv[2]));

    time_t timeEnd = time(0);

    float R = (timeEnd - timeStart);
    printf("%f sec\n",R);;

    return 0;
}
__device__
bool is_prime(int number) {
    if (number % 2 == 0)
        return false;
    float max = sqrtf(number)+1;
    for (int i = 3; i < max; i += 2) {
        if (number % i == 0) {
            return false;
        }
    }
    return true;
}
__global__
void loop_prime(int max,int pas,int * j) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;;
    int i = index;
    if (i%2==0){
      i+=pas+1;
    }
    //printf("%d ",i);
    for (; i < max; i += pas*2) {
        if (is_prime(i)) {
            j[index]++;
        }
    }
    //printf("%d ",j[index]);
}

void go(const int max, const int number_block) {
    int nbBlock = number_block;
    int nbThread = 64;
    int nb = nbBlock * nbThread;
    int j = 0;

    printf("Prime Benchmark : %d\n", max);

    int ha[nb];
    int *da;

    hipMalloc((void **)&da, nb*sizeof(int));

    for (int i = 0; i<nb; ++i) {
        ha[i] = 0;
    }

    hipMemcpy(da, ha, nb*sizeof(int), hipMemcpyHostToDevice);

    loop_prime<<<nbBlock, nbThread>>>(max,nb,da);

    hipMemcpy(ha, da, nb*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<nb; ++i) {
        j+=ha[i];
    }

    hipFree(da);

    printf("\nThere are %d prime numbers between 1 and %d \n",j,max);
}
