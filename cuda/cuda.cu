
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

__device__ bool is_prime(int number);
__global__ void loop_prime(int max,int pas, int * j);
void go(const int max, const int number_thread);

/*
    argv[1] = limit
    argv[2] = Thread number
*/
int main(int argc, char *argv[]) {

    //QCoreApplication a(argc, argv);
    time_t timeStart = time(0);

    go(atoi(argv[1]),atoi(argv[2]));

    time_t timeEnd = time(0);

    float R = (timeEnd - timeStart);
    printf("%f sec\n",R);;

    //return a.exec();
    return 0;
}
__device__
bool is_prime(int number) {
    if (number % 2 == 0)
        return false;
    double max = sqrtf(number)+1;
    for (int i = 3; i < max; i += 2) {
        if (number % i == 0) {
            return false;
        }
    }
    return true;
}
__global__
void loop_prime(int max,int pas,int * j) {
    for (int i = blockIdx.x; i < max; i += pas) {
        if (is_prime(i)) {
            j[blockIdx.x]++;
        }
    }
    //printf("%d ",j[blockIdx.x]);
}

void go(const int max, const int number_thread) {
    int j = 0;
    printf("Prime Benchmark : %d\n", max);

    int ha[number_thread];
    int *da;
    hipMalloc((void **)&da, number_thread*sizeof(int));

    for (int i = 0; i<number_thread; ++i) {
        ha[i] = 0;
    }

    hipMemcpy(da, ha, number_thread*sizeof(int), hipMemcpyHostToDevice);

    loop_prime<<<number_thread, 1>>>(max,number_thread,da);

    hipMemcpy(ha, da, number_thread*sizeof(int), hipMemcpyDeviceToHost);
    
    
    for (int i = 0; i<number_thread; ++i) {
        j+=ha[i];
    }

    hipFree(da);

    printf("\nThere are %d prime numbers between 1 and %d \n",j,max);
}
