
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

using namespace std;

__device__ bool is_prime(int number);
__global__ void loop_prime(int max,int pas, int * j);
void go(const int max, const int number_thread);

/*
    argv[1] = limit
    argv[2] = Thread number
*/
int main(int argc, char *argv[]) {

    //QCoreApplication a(argc, argv);
    time_t timeStart = time(0);

    go(atoi(argv[1]),atoi(argv[2]));

    time_t timeEnd = time(0);

    float R = (timeEnd - timeStart);
    printf("%f sec\n",R);;

    //return a.exec();
    return 0;
}
__device__
bool is_prime(int number) {
    if (number % 2 == 0)
        return false;
    double max = number/2;
    for (int i = 3; i < max; i += 2) {
        if (number % i == 0) {
            return false;
        }
    }
    return true;
}
__global__
void loop_prime(int max,int pas,int * j) {
    int jj=0;
    for (int i = blockIdx.x; i < max; i += pas) {
        if (is_prime(i)) {
            jj++;
        }
    }
    (*j)+=jj;
}

void go(const int max, const int number_thread) {
    int j = 0;
    int ha[number_thread];

    printf("Prime Benchmark : %d\n", max);

    int *da;
    hipMalloc((void **)&da, number_thread*sizeof(int));

    for (int i = 0; i<number_thread; ++i) {
        ha[i] = 0;
    }

    hipMemcpy(da, ha, number_thread*sizeof(int), hipMemcpyHostToDevice);

    loop_prime<<<number_thread, 1>>>(max,number_thread,da);

    hipMemcpy(ha, da, number_thread*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<number_thread; ++i) {
        j+=ha[i];
    }

    hipFree(da);

    printf("There are %d prime numbers between 1 and %d \n",j,max);
}
